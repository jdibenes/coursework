#include "hip/hip_runtime.h"

#include "cudagolgrid.h"

#define BLOCK_WIDTH  32
#define BLOCK_HEIGHT 16

namespace {
template <int MASK_WIDTH, int MASK_HEIGHT>
__global__ void kernelGOLNaive(gol::cell const *current, gol::cell *next, int width, int height) {
    __shared__ gol::cell tile[MASK_HEIGHT][MASK_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = MASK_WIDTH  - 2;
    int by = MASK_HEIGHT - 2;
    int x  = (blockIdx.x * bx) + tx;
    int y  = (blockIdx.y * by) + ty;

    int lx;
    int ly;

    if (x < 1) { lx = x - 1 + width;  } else if (x <= width ) { lx = x - 1; } else { lx = x - 1 - width;  }
    if (y < 1) { ly = y - 1 + height; } else if (y <= height) { ly = y - 1; } else { ly = y - 1 - height; }

    tile[ty][tx] = current[(ly * width) + lx];

    __syncthreads();

    if ((ty >= by) || (tx >= bx)) { return; }

    gol::cell alive = tile[ty    ][tx] + tile[ty    ][tx + 1] + tile[ty    ][tx + 2] +
                      tile[ty + 1][tx] + 8                    + tile[ty + 1][tx + 2] +
                      tile[ty + 2][tx] + tile[ty + 2][tx + 1] + tile[ty + 2][tx + 2];

    int state;

    switch (alive) {
    case 2:  state = tile[ty + 1][tx + 1]; break;
    case 3:  state = GOL_STATE_ALIVE;      break;
    default: state = GOL_STATE_DEAD;
    }

    if (y < height && x < width) { next[(y * width) + x] = state; }
}
}

namespace cudagol {
grid::grid() {
}

grid::~grid() {
}

void grid::create(int width, int height) {
    m_width  = width;
    m_height = height;
    m_bytes  = width * height * sizeof(gol::cell);

    hipMalloc(&m_d_current, m_bytes);
    hipMalloc(&m_d_next,    m_bytes);

    m_grid  = dim3(ceil(width / (float)(BLOCK_WIDTH - 2)), ceil(height / (float)(BLOCK_HEIGHT - 2)));
    m_block = dim3(BLOCK_WIDTH, BLOCK_HEIGHT);
}

void grid::destroy() {
    hipFree(m_d_next);
    hipFree(m_d_current);
}

void grid::read(gol::cell *data) const {
    hipMemcpy(data, m_d_current, m_bytes, hipMemcpyDeviceToHost);
}

void grid::write(gol::cell const *data) const {
    hipMemcpy(m_d_current, data, m_bytes, hipMemcpyHostToDevice);
}

void grid::swapBuffers() {
    gol::cell *tmp = m_d_current;
    m_d_current = m_d_next;
    m_d_next = tmp;
}

void grid::singleStep() {
    kernelGOLNaive<BLOCK_WIDTH, BLOCK_HEIGHT><<<m_grid, m_block>>>(m_d_current, m_d_next, m_width, m_height);
    swapBuffers();
}

int grid::width() const {
    return m_width;
}

int grid::height() const {
    return m_height;
}

int grid::bytes() const {
    return m_bytes;
}

gol::cell *grid::d_current() const {
    return m_d_current;
}

gol::cell *grid::d_next() const {
    return m_d_next;
}
}
