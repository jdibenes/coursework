#include "hip/hip_runtime.h"

#include "cudagolgridgl.h"
#include "utilities.h"

#define MASK_WIDTH  32
#define MASK_HEIGHT 16

namespace {
__global__ void kernelGOLNaive(hipSurfaceObject_t current, hipSurfaceObject_t next, int width, int height, int cols, int shr, int mask) {
    __shared__ gol::cell tile[MASK_HEIGHT][MASK_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = MASK_WIDTH  - 2;
    int by = MASK_HEIGHT - 2;
    int x  = (blockIdx.x * bx) + tx;
    int y  = (blockIdx.y * by) + ty;

    int lx;
    int ly;

    if (x < 1) { lx = x - 1 + width;  } else if (x <= width ) { lx = x - 1; } else { lx = x - 1 - width;  }
    if (y < 1) { ly = y - 1 + height; } else if (y <= height) { ly = y - 1; } else { ly = y - 1 - height; }

    surf3Dread(&tile[ty][tx], current, lx & mask, ly & mask, ((ly >> shr) * cols) + (lx >> shr));

    __syncthreads();

    if ((ty >= by) || (tx >= bx)) { return; }

    gol::cell alive = tile[ty    ][tx] + tile[ty    ][tx + 1] + tile[ty    ][tx + 2] +
                      tile[ty + 1][tx] + 8                    + tile[ty + 1][tx + 2] +
                      tile[ty + 2][tx] + tile[ty + 2][tx + 1] + tile[ty + 2][tx + 2];

    int state;

    switch (alive) {
    case 2:  state = tile[ty + 1][tx + 1]; break;
    case 3:  state = GOL_STATE_ALIVE;      break;
    default: state = GOL_STATE_DEAD;
    }

    if (y < height && x < width) { surf3Dwrite((gol::cell)state, next, x & mask, y & mask, ((y >> shr) * cols) + (x >> shr)); }
}
}

namespace cudagol {
gridGL::gridGL() {
}

gridGL::~gridGL() {
}

void gridGL::setDimensions(int width, int height) {
    m_width = width;
    m_height = height;

    m_grid  = dim3(ceil(width / (float)(MASK_WIDTH - 2)), ceil(height / (float)(MASK_HEIGHT - 2)));
    m_block = dim3(MASK_WIDTH, MASK_HEIGHT);
}

void gridGL::registerTextures(int TILE_L, int cols, GLuint tex1, GLuint tex2) {
    m_cols = cols;
    utilities::divPOTshrmask(TILE_L, m_shr, m_mask);

    hipGraphicsGLRegisterImage(&m_tex[0], tex1, GL_TEXTURE_3D, cudaGraphicsMapFlagsNone);
    hipGraphicsGLRegisterImage(&m_tex[1], tex2, GL_TEXTURE_3D, cudaGraphicsMapFlagsNone);
}

void gridGL::unregisterTextures() {
    hipGraphicsUnregisterResource(m_tex[1]);
    hipGraphicsUnregisterResource(m_tex[0]);
}

void gridGL::createSurface(hipGraphicsResource_t tex, hipSurfaceObject_t *surf) {
    hipArray_t cudaArr;
    hipResourceDesc cudaRD;

    hipGraphicsSubResourceGetMappedArray(&cudaArr, tex, 0, 0);
    cudaRD.resType = hipResourceTypeArray;
    cudaRD.res.array.array = cudaArr;
    hipCreateSurfaceObject(surf, &cudaRD);
}

void gridGL::createSurfaces() {
    hipGraphicsMapResources(2, m_tex);
    createSurface(m_tex[0], &m_surf[0]);
    createSurface(m_tex[1], &m_surf[1]);
}

void gridGL::deleteSurfaces() {
    hipDestroySurfaceObject(m_surf[1]);
    hipDestroySurfaceObject(m_surf[0]);
    hipGraphicsUnmapResources(2, m_tex);
}

void gridGL::setFlags() {
    cudaGraphicsResourceSetMapFlags(m_tex[0], cudaGraphicsMapFlagsReadOnly);
    cudaGraphicsResourceSetMapFlags(m_tex[1], cudaGraphicsMapFlagsWriteDiscard);
}

void gridGL::swapBuffers() {
    hipGraphicsResource_t tmp = m_tex[0];
    m_tex[0] = m_tex[1];
    m_tex[1] = tmp;
}

void gridGL::singleStepRender() {
    setFlags();
    createSurfaces();
    kernelGOLNaive<<<m_grid, m_block>>>(m_surf[0], m_surf[1], m_width, m_height, m_cols, m_shr, m_mask);
    deleteSurfaces();
    swapBuffers();
}
}
