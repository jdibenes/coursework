#include "hip/hip_runtime.h"

#include "cudagolrendergl.h"
#include "utilities.h"

namespace {
template<typename T>
__global__ void kernelGOLRenderGL(gol::cell const *current, hipSurfaceObject_t surface, int cols, int width, int height, int shr, int mask) {
    int x = ((blockIdx.x * blockDim.x) + threadIdx.x) * sizeof(T);
    int y =  (blockIdx.y * blockDim.y) + threadIdx.y;

    if (y < height && x < width) { surf3Dwrite(*(T *)(current + (y * width) + x), surface, x & mask, y & mask, ((y >> shr) * cols) + (x >> shr)); }
}
}

namespace cudagol {
RenderGL::RenderGL() {
}

RenderGL::~RenderGL() {
}

void RenderGL::setGrid(int width, int height, int BLOCK_WIDTH, int BLOCK_HEIGHT) {
    m_width  = width;
    m_height = height;

    switch (width & 3) {
    case 0:  m_mode = 2; break;
    case 1:  m_mode = 0; break;
    case 2:  m_mode = 1; break;
    default: m_mode = 0;
    }

    m_grid  = dim3(ceil(width  / (double)(BLOCK_WIDTH << m_mode)), ceil(height / (double)BLOCK_HEIGHT));
    m_block = dim3(BLOCK_WIDTH, BLOCK_HEIGHT);
}

void RenderGL::registerTexture(int TILE_L, int cols, GLuint tex) {
    m_cols = cols;
    utilities::divPOTshrmask(TILE_L, m_shr, m_mask);

    hipGraphicsGLRegisterImage(&m_tex, tex, GL_TEXTURE_3D, cudaGraphicsMapFlagsWriteDiscard);
}

void RenderGL::unregisterTexture() {
    hipGraphicsUnregisterResource(m_tex);
}

void RenderGL::createSurface() {
    hipGraphicsMapResources(1, &m_tex);
    hipArray_t cudaArr;
    hipGraphicsSubResourceGetMappedArray(&cudaArr, m_tex, 0, 0);
    hipResourceDesc cudaRD;
    cudaRD.resType = hipResourceTypeArray;
    cudaRD.res.array.array = cudaArr;
    hipCreateSurfaceObject(&m_surf, &cudaRD);
}

void RenderGL::deleteSurface() {
    hipDestroySurfaceObject(m_surf);
    hipGraphicsUnmapResources(1, &m_tex);
}

void RenderGL::render(gol::cell *d_current) {
    createSurface();
    switch (m_mode) {
    case 0: kernelGOLRenderGL<u8 ><<<m_grid, m_block>>>(d_current, m_surf, m_cols, m_width, m_height, m_shr, m_mask); break;
    case 1: kernelGOLRenderGL<u16><<<m_grid, m_block>>>(d_current, m_surf, m_cols, m_width, m_height, m_shr, m_mask); break;
    case 2: kernelGOLRenderGL<u32><<<m_grid, m_block>>>(d_current, m_surf, m_cols, m_width, m_height, m_shr, m_mask); break;
    }
    deleteSurface();
}

int RenderGL::width() const {
    return m_width;
}

int RenderGL::height() const {
    return m_height;
}

int RenderGL::cols() const {
    return m_cols;
}

int RenderGL::shr() const {
    return m_shr;
}

int RenderGL::mask() const {
    return m_mask;
}

int RenderGL::mode() const {
    return m_mode;
}
}
