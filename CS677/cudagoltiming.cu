
#include "cudagoltiming.h"

namespace cudagol {
timing::timing() {
    hipEventCreate(&m_start);
    hipEventCreate(&m_stop);
}

timing::~timing() {
    hipEventDestroy(m_start);
    hipEventDestroy(m_stop);
}

void timing::start() {
    hipEventRecord(m_start);
    //hipEventSynchronize(m_start);
}

void timing::stop() {
    hipEventRecord(m_stop);
    hipEventSynchronize(m_stop);
}

float timing::ms() {
    float ms;
    hipEventElapsedTime(&ms, m_start, m_stop);
    return ms;
}

void timing::sync() {
    hipDeviceSynchronize();
}
}
